#include "hip/hip_runtime.h"
// Copyright (c) 2010-2015, Raymond Tay, Singapore
// All rights reserved.
// 
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 
//     * Redistributions of source code must retain the above copyright
//       notice, this list of conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright
//       notice, this list of conditions and the following disclaimer in the
//       documentation and/or other materials provided with the distribution.
//     * Neither the name of the <organization> nor the
//       names of its contributors may be used to endorse or promote products
//       derived from this software without specific prior written permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
// ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
// WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
// DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
// (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
// LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
// ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
// (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//

#ifndef _TEMPLATE_KERNEL_H_
#define _TEMPLATE_KERNEL_H_

#include <stdio.h>
#include "reduction.h"

#define SDATA( index)      cutilBankChecker(sdata, index)

////////////////////////////////////////////////////////////////////////////////
//! Pearson Coefficient Kernel in Python 
////////////////////////////////////////////////////////////////////////////////
//def pearson(v1, v2):
//	sum1=sum(v1)
//	sum2=sum(v2)
//
//	sum1sq=sum([pow(v,2) for v in v1])
//	sum2sq=sum([pow(v,2) for v in v2])
//
//	psum=sum([v1[i]*v2[i] for i in range(len(v1))])
//	num=psum-(sum1*sum2/len(v1))
//	den=sqrt((sum1sq-pow(sum1,2)/len(v1))*(sum2sq-pow(sum2,2)/len(v1)))
//	if den==0: return 0
//	return 1.0-num/den
//
////////////////////////////////////////////////////////////////////////////////
//! Pearson Coefficient Kernel in CUDA 
////////////////////////////////////////////////////////////////////////////////
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
#define DATA_SIZE 256

__global__ void
testKernel( float* g_v1data,float* g_v2data, float* g_odata) 
{
  // shared memory
  __shared__  float sdata[DATA_SIZE];

  // access thread id
  const unsigned int tid = threadIdx.x;
  // access number of threads in this block
  const unsigned int num_threads = blockDim.x;

  // read in input data from global memory
  // use the bank checker macro to check for bank conflicts during host
  // emulation
  SDATA(tid) = g_v1data[tid];
  __syncthreads();

  // perform some computations
  SDATA(tid) = (float) num_threads * SDATA( tid);
  __syncthreads();

  // write data to global memory
  g_odata[tid] = SDATA(tid);
}

#endif // #ifndef _TEMPLATE_KERNEL_H_
